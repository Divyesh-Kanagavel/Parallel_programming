#include <stdio.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* funcName, const char* fileName, int LineNum)
{
  if (err!=hipSuccess)
  {
    fprintf(stderr, "CUDA error at %s:%d code = %d(%s) \"%s\" \n", fileName, LineNum, static_cast<int>(err), hipGetErrorString(err), funcName);
    exit(EXIT_FAILURE);
  }
}

__global__ void vectorAdd(const float *A, const float *B, float *C, int NumElements)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < NumElements)
  {
    C[i] = A[i] + B[i];
  }
}

int main()
{
  int NumElements = 50000;
  size_t size = NumElements * sizeof(float);
  float *h_a, *h_b, *h_c; // buffers in host
  float *d_a, *d_b, *d_c; // buffers in device
  hipStream_t stream1, stream2;

  h_a = (float *)malloc(size);
  h_b = (float *)malloc(size);
  h_c = (float *)malloc(size);

  for(int i=0;i<NumElements;i++)
  {
    h_a[i] = rand() / (float)RAND_MAX;
    h_b[i] = rand() / (float)RAND_MAX;
  }

  CHECK_CUDA_ERROR(hipMalloc(&d_a, size));
  CHECK_CUDA_ERROR(hipMalloc(&d_b, size));
  CHECK_CUDA_ERROR(hipMalloc(&d_c, size));
  CHECK_CUDA_ERROR(hipStreamCreate(&stream1));
  CHECK_CUDA_ERROR(hipStreamCreate(&stream2));

  hipEvent_t event;
  CHECK_CUDA_ERROR(hipEventCreate(&event));
  

  // memcpy from host to device in two cuda_streams - the control is passed back to cpu while the mem copy happens utilizing the gpus in the background

  CHECK_CUDA_ERROR(hipMemcpyAsync(d_a, h_a, size, hipMemcpyHostToDevice,stream1));
  CHECK_CUDA_ERROR(hipMemcpyAsync(d_b, h_b, size, hipMemcpyHostToDevice,stream2));

  CHECK_CUDA_ERROR(hipEventRecord(event,stream2));

  CHECK_CUDA_ERROR(hipStreamWaitEvent(stream1, event,0));

  int threadsPerBlock = 256;
  int numBlocks = (NumElements + threadsPerBlock - 1) / threadsPerBlock;

  vectorAdd<<<numBlocks, threadsPerBlock, 0, stream1>>>(d_a, d_b, d_c, NumElements);


  CHECK_CUDA_ERROR(hipMemcpyAsync(h_c, d_c, size,hipMemcpyDeviceToHost, stream1));

  CHECK_CUDA_ERROR(hipStreamSynchronize(stream1));
  CHECK_CUDA_ERROR(hipStreamSynchronize(stream2));

  free(h_a);
  free(h_b);
  free(h_c);
  CHECK_CUDA_ERROR(hipFree(d_a));
  CHECK_CUDA_ERROR(hipFree(d_b));
  CHECK_CUDA_ERROR(hipFree(d_c));
  CHECK_CUDA_ERROR(hipStreamDestroy(stream1));
  CHECK_CUDA_ERROR(hipStreamDestroy(stream2));
  CHECK_CUDA_ERROR(hipEventDestroy(event));

  
  return 0;
}