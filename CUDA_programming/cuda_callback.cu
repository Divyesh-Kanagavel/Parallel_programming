#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
template <typename T>
void check(T err, const char* const func, const char* const file, const int line) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line, static_cast<unsigned int>(err), hipGetErrorString(err), func);
        exit(EXIT_FAILURE);
    }
}

// kernel1
__global__ void kernel1(float *data, int n)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
  {
    data[i] *= 2.0f;
  }
}

// kernel 2
__global__ void kernel2(float *data, int n)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
  {
    data[i] += 1.0f;
  }
}

void CUDART_CB myStreamCallBack(hipStream_t stream, hipError_t status, void *userData)
{
  printf("Stream callback : Operation completed!\n");
}



int main()
{
  const int N = 1000000;
  size_t size = N * sizeof(float);
  float *h_a, *d_a;

  hipStream_t stream1, stream2;
  hipEvent_t event;

  CHECK_CUDA_ERROR(hipHostMalloc(&h_a, size, hipHostMallocDefault)); //pinned memory for faster transfers
  CHECK_CUDA_ERROR(hipMalloc(&d_a, size)); // memory allocation in device



  for(int i=0;i<N;i++)
  {
    h_a[i] = static_cast<float>(i);
  }

  int leastPriority, greatestPriority;
  CHECK_CUDA_ERROR(hipDeviceGetStreamPriorityRange(&leastPriority, &greatestPriority));

  CHECK_CUDA_ERROR(hipStreamCreateWithPriority(&stream1, hipStreamNonBlocking, leastPriority));
  CHECK_CUDA_ERROR(hipStreamCreateWithPriority(&stream2, hipStreamNonBlocking, greatestPriority));


  CHECK_CUDA_ERROR(hipEventCreate(&event));

  CHECK_CUDA_ERROR(hipMemcpyAsync(d_a, h_a, size, hipMemcpyHostToDevice, stream1));

  kernel1<<<(N + 255)/256, 256, 0, stream1>>>(d_a, N);
  
  CHECK_CUDA_ERROR(hipEventRecord(event, stream1));

  CHECK_CUDA_ERROR(hipStreamWaitEvent(stream2, event, 0));

  kernel2<<<(N + 255)/256, 256, 0, stream2>>>(d_a, N);

  // add callback to stream2
  CHECK_CUDA_ERROR(hipStreamAddCallback(stream2, myStreamCallBack, NULL, 0));

  CHECK_CUDA_ERROR(hipMemcpyAsync(h_a, d_a, size, hipMemcpyDeviceToHost, stream2));

  CHECK_CUDA_ERROR(hipStreamSynchronize(stream1));
  CHECK_CUDA_ERROR(hipStreamSynchronize(stream2));

  // verification of results

  for(int i=0;i<N;i++)
  {
    float expected = 2.0f * static_cast<float>(i) + 1.0f;
    if (fabs(h_a[i]-expected) > 1e-5)
    {
      fprintf(stderr, "Result verifiation failed!\n");
      exit(EXIT_FAILURE);

    }
  }
  printf("Results verified!\n");

  return 0;
}